#include "hip/hip_runtime.h"
#include "Postprocessing.cuh"
#include <algorithm>


int MaskSobelX[9] = {
	-1, 0, 1,
	-2, 0, 2,
	-1, 0, 1
};

int MaskSobelY[9] = {
	1, 2, 1,
	0, 0, 0,
	-1, -2, -1
};

int* gpu_maskSobel_x;
int* gpu_maskSobel_y;

uchar3* pixelBuffer;

uchar3* gpu_srcBuffer;
uchar3* gpu_outBuffer;

uchar3* gpu_tempBuffer1;

__global__ void copy_image(int _width, uchar3* _srcData, uchar3* _outData)
{
	//��ǥ�� �ľ�
	int xPos = blockIdx.x * blockDim.x + threadIdx.x;
	int yPos = blockIdx.y * blockDim.y + threadIdx.y;


	_outData[yPos * _width + xPos] = _srcData[yPos * _width + xPos];
}

__global__ void gpu_gray(int _width, uchar3* _srcData, uchar3* _outData)
{
	//��ǥ�� �ľ�
	int xPos = blockIdx.x * blockDim.x + threadIdx.x;
	int yPos = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned char brightness = (_srcData[yPos * _width + xPos].x + _srcData[yPos * _width + xPos].y + _srcData[yPos * _width + xPos].z) / 3;
	_outData[yPos * _width + xPos] = { brightness ,brightness ,brightness };
}

__global__ void calculate_sobel_gpu(int _width, uchar3* _srcData, uchar3* _outData, int* _maskSobelX, int* _maskSobelY)
{
	//��ǥ�� �ľ�
	int xPos = blockIdx.x * blockDim.x + threadIdx.x;
	int yPos = blockIdx.y * blockDim.y + threadIdx.y;

	uchar3 gradientX = {0,0,0};
	uchar3 gradientY = { 0,0,0 };
	//���� �ȼ��� ��ġ (xPos,yPos)
	for (int k = 0; k < 9; k++)
	{
		int r = k / 3; //k = 0,1,2 �� r = 0 , k = 3,4,5 �� r =1
		int c = k % 3;
		// k = 0 �� �� i,j = (x - 1, y - 1)
		// k = 1 �� �� i,j = (x, y - 1)
		// k = 0 �� �� (r,c ) = (0,0) (r-1, c - 1) = (-1, -1)
		// k = 1 �� �� (r, c) = (0,1) (r-1, c-1)	= (-1,0)
		// k = 2 �� �� (r, c) = (0, 2) (r-1, c-1) = (-1, 1)
		int idx = (yPos + r - 1) * _width + (xPos + c - 1);
		gradientX.x = gradientX.x + _maskSobelX[k] * _srcData[idx].x;
		gradientY.x = gradientY.x + _maskSobelY[k] * _srcData[idx].x;

		gradientX.y = gradientX.y + _maskSobelX[k] * _srcData[idx].y;
		gradientY.y = gradientY.y + _maskSobelY[k] * _srcData[idx].y;

		gradientX.z = gradientX.z + _maskSobelX[k] * _srcData[idx].z;
		gradientY.z = gradientY.z + _maskSobelY[k] * _srcData[idx].z;
	}
	uchar3 magnitude;
	magnitude.x = sqrtf(gradientX.x * gradientX.x + gradientY.x * gradientY.x);
	magnitude.y = sqrtf(gradientX.y * gradientX.y + gradientY.y * gradientY.y);
	magnitude.z = sqrtf(gradientX.z * gradientX.z + gradientY.z * gradientY.z);

	_outData[yPos * WIDTH + xPos] = magnitude;
}


__global__ void calculate_sobel2_gpu(int _width, uchar3* _srcData, uchar3* _outData, uchar3 _min, uchar3 _max)
{
	//��ǥ�� �ľ�
	int xPos = blockIdx.x * blockDim.x + threadIdx.x;
	int yPos = blockIdx.y * blockDim.y + threadIdx.y;


	int currentIndex = _width * yPos + xPos;

	uchar3 newPixel;
	{
		float g = _srcData[yPos * _width + xPos].x;
		float t = (g - _min.x) / (_max.x - _min.x);
		newPixel.x = static_cast<unsigned char>(t * 255);
	}
	{
		float g = _srcData[yPos * _width + xPos].y;
		float t = (g - _min.y) / (_max.y - _min.y);
		newPixel.y = static_cast<unsigned char>(t * 255);
	}
	{
		float g = _srcData[yPos * _width + xPos].z;
		float t = (g - _min.z) / (_max.z - _min.z);
		newPixel.z = static_cast<unsigned char>(t * 255);
	}

	if ((newPixel.x + newPixel.y + newPixel.z)/3 < 160)
	{
		newPixel = { 0,0,0 };
	}
	unsigned char brightness = (newPixel.x + newPixel.y + newPixel.z) / 3;
	_outData[yPos * WIDTH + xPos] = { brightness  ,brightness ,brightness };
}



std::vector<std::vector<std::function<void(uchar3* _targetData, uchar3* _desData)>>> Postprocessing::postprocessingFunc;

void Postprocessing::init()
{
	hipError_t Status = hipSetDevice(0);

	//�迭 �޸� �Ҵ�
	pixelBuffer = new uchar3[WIDTH * HEIGHT];

	 Status = hipMalloc((void**)&gpu_maskSobel_x, sizeof(int) * 9);
	assert(Status == hipSuccess);
	Status = hipMalloc((void**)&gpu_maskSobel_y, sizeof(int) * 9);
	assert(Status == hipSuccess);
	Status = hipMalloc((void**)&gpu_srcBuffer, sizeof(uchar3) * WIDTH * HEIGHT);
	assert(Status == hipSuccess);
	Status = hipMalloc((void**)&gpu_outBuffer, sizeof(uchar3) * WIDTH * HEIGHT);
	assert(Status == hipSuccess);
	Status = hipMalloc((void**)&gpu_tempBuffer1, sizeof(uchar3) * WIDTH * HEIGHT);
	assert(Status == hipSuccess);

	Status = hipMemcpy(gpu_maskSobel_x, MaskSobelX, sizeof(int) * 9, hipMemcpyHostToDevice);
	assert(Status == hipSuccess);
	Status = hipMemcpy(gpu_maskSobel_y, MaskSobelY, sizeof(int) * 9, hipMemcpyHostToDevice);
	assert(Status == hipSuccess);

	postprocessingFunc.resize(static_cast<int>(OperationMode::Max));
	for (int i = 0; i < postprocessingFunc.size(); i++)
	{
		postprocessingFunc[i].resize(static_cast<int>(Filter::Max));
	}

	postprocessingFunc[static_cast<int>(OperationMode::GPU)][static_cast<int>(Filter::Sobel)] = std::bind([=](uchar3* _targetData, uchar3* _desData)
		{
			hipError_t Status = hipMemcpy(gpu_srcBuffer, _targetData, sizeof(uchar3) * WIDTH * HEIGHT, hipMemcpyHostToDevice);
			assert(Status == hipSuccess);
			dim3 gridDim(32, 24, 1);
			dim3 blockDim(32, 32, 1);
			calculate_sobel_gpu << <gridDim, blockDim >> > (WIDTH, gpu_srcBuffer, gpu_tempBuffer1, gpu_maskSobel_x, gpu_maskSobel_y);
			hipDeviceSynchronize();

			Status = hipMemcpy(_desData, gpu_tempBuffer1, sizeof(uchar3) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);
			assert(Status == hipSuccess);

			uchar3 gradientMin = *std::min_element(_desData, _desData + WIDTH * HEIGHT - 1,[](uchar3 _a, uchar3 _b)
				{
					return (_a.x + _a.y + _a.z) < (_b.x + _b.y + _b.z);
				});
			uchar3 gradientMax = *std::max_element(_desData, _desData + WIDTH * HEIGHT - 1, [](uchar3 _a, uchar3 _b)
				{
					return (_a.x + _a.y + _a.z) < (_b.x + _b.y + _b.z);
				});
			calculate_sobel2_gpu<<<gridDim, blockDim >>>(WIDTH, gpu_tempBuffer1, gpu_outBuffer, gradientMin, gradientMax);
			hipDeviceSynchronize();

			Status = hipMemcpy(_desData, gpu_outBuffer, sizeof(uchar3) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);
			assert(Status == hipSuccess);

		},
		std::placeholders::_1, std::placeholders::_2);

	postprocessingFunc[static_cast<int>(OperationMode::CPU)][static_cast<int>(Filter::Sobel)] = std::bind([=](uchar3* _targetData, uchar3* _desData)
		{
			//�׷����Ʈ ũ�� ���ϱ�
			for (int y = 1; y < HEIGHT - 1; y++)
			{
				for (int x = 1; x < WIDTH - 1; x++)
				{
					uchar3 gradientX = {0,0,0};
					uchar3 gradientY = {0,0,0};
					//���� �ȼ��� ��ġ (x,y)
					for (int k = 0; k < 9; k++)
					{
						int r = k / 3; //k = 0,1,2 �� r = 0 , k = 3,4,5 �� r =1
						int c = k % 3;
						// k = 0 �� �� i,j = (x - 1, y - 1)
						// k = 1 �� �� i,j = (x, y - 1)
						// k = 0 �� �� (r,c ) = (0,0) (r-1, c - 1) = (-1, -1)
						// k = 1 �� �� (r, c) = (0,1) (r-1, c-1)	= (-1,0)
						// k = 2 �� �� (r, c) = (0, 2) (r-1, c-1) = (-1, 1)
						int idx = (y + r - 1) * WIDTH + (x + c - 1);


						gradientX.x = gradientX.x + MaskSobelX[k] * _targetData[idx].x;
						gradientY.x = gradientY.x + MaskSobelY[k] * _targetData[idx].x;

						gradientX.y = gradientX.y + MaskSobelX[k] * _targetData[idx].y;
						gradientY.y = gradientY.y + MaskSobelY[k] * _targetData[idx].y;

						gradientX.z = gradientX.z + MaskSobelX[k] * _targetData[idx].z;
						gradientY.z = gradientY.z + MaskSobelY[k] * _targetData[idx].z;
					}
					uchar3 magnitude;
					magnitude.x = sqrtf(gradientX.x * gradientX.x + gradientY.x * gradientY.x);
					magnitude.y = sqrtf(gradientX.y * gradientX.y + gradientY.y * gradientY.y);
					magnitude.z = sqrtf(gradientX.z * gradientX.z + gradientY.z * gradientY.z);
					//float magnitude = sqrtf(Gx * Gx + Gy * Gy);
					pixelBuffer[y * WIDTH + x] = magnitude;
				}
			}

			////��� �̹��� ����
			//for (int y = 1; y < HEIGHT - 1; y++)
			//{
			//	for (int x = 1; x < WIDTH - 1; x++)
			//	{
			//		_desData[y * WIDTH + x] = pixelBuffer[y * WIDTH + x];
			//	}
			//}

			//�׷����Ʈ ũ���� �ִ밪�� �ּڰ� ���ϱ�
			uchar3 min = { 255,255,255 };
			uchar3 max = { 0,0,0 };

			for (int y = 1; y < HEIGHT - 1; y++)
			{
				for (int x = 1; x < WIDTH - 1; x++)
				{
					int idx = y * WIDTH + x;

					int buffer_brightness = (pixelBuffer[idx].x + pixelBuffer[idx].y + pixelBuffer[idx].z) / 3;
					int min_brightness = (min.x + min.y + min.z) / 3;
					int max_brightness = (max.x + max.y + max.z) / 3;

					if (buffer_brightness < min_brightness)
						min = pixelBuffer[idx];
					if (buffer_brightness > max_brightness)
						max = pixelBuffer[idx];
				}
			}

			//��� �̹��� ����
			for (int y = 1; y < HEIGHT - 1; y++)
			{
				for (int x = 1; x < WIDTH - 1; x++)
				{
					uchar3 newPixel;
					{
						float g = pixelBuffer[y * WIDTH + x].x;
						float t = (g - min.x) / (max.x - min.x);
						newPixel.x = static_cast< unsigned char>((t ) * 255);
					}
					{
						float g = pixelBuffer[y * WIDTH + x].y;
						float t = (g - min.y) / (max.y - min.y);
						newPixel.y = static_cast<unsigned char>((t ) * 255);
					}
					{
						float g = pixelBuffer[y * WIDTH + x].z;
						float t = (g - min.z) / (max.z - min.z);
						newPixel.z = static_cast<unsigned char>((t ) * 255);
					}

					int brightness = (newPixel.x + newPixel.y + newPixel.z) / 3;
					if (brightness < 160)
					{
						brightness = 0;
					}

					_desData[y * WIDTH + x] = { (unsigned char)brightness,(unsigned char)brightness ,(unsigned char)brightness };
				}
			}
		},
		std::placeholders::_1, std::placeholders::_2);

	postprocessingFunc[static_cast<int>(OperationMode::CPU)][static_cast<int>(Filter::None)] = std::bind([=](uchar3* _targetData, uchar3* _desData)
		{
			//��� �̹��� ����
			for (int y = 1; y < HEIGHT - 1; y++)
			{
				for (int x = 1; x < WIDTH - 1; x++)
				{
					_desData[y * WIDTH + x] = _targetData[y * WIDTH + x];
				}
			}
		}
	, std::placeholders::_1, std::placeholders::_2);

	postprocessingFunc[static_cast<int>(OperationMode::GPU)][static_cast<int>(Filter::None)] = std::bind([=](uchar3* _targetData, uchar3* _desData)
		{
			hipError_t Status = hipMemcpy(gpu_srcBuffer, _targetData, sizeof(uchar3) * WIDTH * HEIGHT, hipMemcpyHostToDevice);
			assert(Status == hipSuccess);

			dim3 gridDim(32,24,1);
			dim3 blockDim(32, 32, 1);
			copy_image << <gridDim, blockDim >> > (WIDTH, gpu_srcBuffer, gpu_outBuffer);
			hipDeviceSynchronize();
			Status = hipMemcpy(_desData, gpu_outBuffer, sizeof(uchar3) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);
			assert(Status == hipSuccess);


		}
	, std::placeholders::_1, std::placeholders::_2);

	postprocessingFunc[static_cast<int>(OperationMode::CPU)][static_cast<int>(Filter::Gray)] = std::bind([=](uchar3* _targetData, uchar3* _desData)
		{
			//��� �̹��� ����
			for (int y = 1; y < HEIGHT - 1; y++)
			{
				for (int x = 1; x < WIDTH - 1; x++)
				{
					unsigned char brightness = (_targetData[y * WIDTH + x].x + _targetData[y * WIDTH + x].y + _targetData[y * WIDTH + x].z) / 3;
					_desData[y * WIDTH + x] = { brightness ,brightness ,brightness };
				}
			}
		}
	, std::placeholders::_1, std::placeholders::_2);

	postprocessingFunc[static_cast<int>(OperationMode::GPU)][static_cast<int>(Filter::Gray)] = std::bind([=](uchar3* _targetData, uchar3* _desData)
		{
			hipError_t Status = hipMemcpy(gpu_srcBuffer, _targetData, sizeof(uchar3) * WIDTH * HEIGHT, hipMemcpyHostToDevice);
			assert(Status == hipSuccess);

			dim3 gridDim(32, 24, 1);
			dim3 blockDim(32, 32, 1);
			gpu_gray << <gridDim, blockDim >> > (WIDTH, gpu_srcBuffer, gpu_outBuffer);
			hipDeviceSynchronize();
			Status = hipMemcpy(_desData, gpu_outBuffer, sizeof(uchar3) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);
			assert(Status == hipSuccess);


		}
	, std::placeholders::_1, std::placeholders::_2);
}

void Postprocessing::release()
{
	delete[] pixelBuffer;
}

void Postprocessing::set_postprocessing(uchar3* _targetData, uchar3* _desData, OperationMode _mode, Filter _filter)
{
	//���� �ʱ�ȭ
	memset(pixelBuffer, 0, sizeof(uchar3) * WIDTH * HEIGHT);

	postprocessingFunc[static_cast<int>(_mode)][static_cast<int>(_filter)](_targetData, _desData);
	

}
